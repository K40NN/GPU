#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdlib.h>
#include <string.h>

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );
    res->m = (double *) calloc(columns * rows, sizeof(double));
    res->columns = columns;
    res->rows = rows;
    res->size = rows * columns * sizeof(double);
    return res;
}

void destroy_matrix(matrix_t *m)
{
    //printf("free %p %p\n", m, m->m);
    free(m->m);
    free(m);
}

void print_matrix(matrix_t *m, bool is_short){
    unsigned lim_rows = 0;
    unsigned lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row ++)
    {
        for (int col = 0; col < lim_col; col ++)
        {
            printf("%.2lf ", m->m[col + row * m->columns]);
        }
        if (is_short && lim_col != m->columns) printf("...");
        printf("\n");
    }
    if (is_short && lim_rows != m->rows) printf("...\n");
}

void hadamard_product(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
            res->m[idx] = m1->m[idx] * m2->m[idx];
    }
}

void matrix_sum(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    { 
        res->m[idx] = m1->m[idx] + m2->m[idx];
    }
}

void matrix_minus(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));
             
    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = m1->m[idx] - m2->m[idx];
    }
}

__global__ void computeMatrixMulGPU(double *A, double *B, double *C, int numARows, int numAColumns, int numBRows, int numBColumns) 
{
    int indexx = blockDim.x * blockIdx.x + threadIdx.x;
    int indexy = blockDim.y * blockIdx.y + threadIdx.y;

    if (indexx < numBColumns && indexy < numARows) {
        double sum = 0.0;
        for (int k = 0; k < numAColumns; k++) {
            sum += A[indexy * numAColumns + k] * B[k * numBColumns + indexx];
        }
        C[indexy * numBColumns + indexx] = sum;
    }
}

void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert((m1->columns == m2->rows) &&
           (m1->rows == res->rows) &&
           (m2->columns == res->columns));

    double *deviceA, *deviceB, *deviceC;

    // Allocate memory on the GPU
    hipMalloc((void **)&deviceA, m1->rows * m1->columns * sizeof(double));
    hipMalloc((void **)&deviceB, m2->rows * m2->columns * sizeof(double));
    hipMalloc((void **)&deviceC, res->rows * res->columns * sizeof(double));

    // Copy data from host to device
    hipMemcpy(deviceA, m1->m, m1->rows * m1->columns * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, m2->m, m2->rows * m2->columns * sizeof(double), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((res->columns + blockDim.x - 1) / blockDim.x, (res->rows + blockDim.y - 1) / blockDim.y);

    // Launch the kernel
    computeMatrixMulGPU<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, m1->rows, m1->columns, m2->rows, m2->columns);

    // Copy the result back to the host
    hipMemcpy(res->m, deviceC, res->rows * res->columns * sizeof(double), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void matrix_function(matrix_t *m1, double (*f)(double), matrix_t *res)
{
    assert ( (m1->columns == res->columns) &&             
             (m1->rows == res->rows));

    for (int idx = 0; idx < m1->rows * m1->columns; idx ++)
    {
        res->m[idx] = f(m1->m[idx]);
    }
}

void matrix_transpose(matrix_t *m1, matrix_t *res)
{
    assert ( (m1->columns == res->rows) &&             
             (m1->rows == res->columns));
    
    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m1->columns; col ++)
        {
            res->m[row + col * m1->rows] = m1->m[col + row * m1->columns];
        }
    }
}

void matrix_scalar(matrix_t *m1, double s, matrix_t *res)
{
    assert ( (m1->rows == res->rows) &&             
             (m1->columns == res->columns));

    for (int idx = 0; idx < m1->columns*m1->rows; idx ++)
    {
        res->m[idx] = m1->m[idx] * s;
    }
}

void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert ( (dest->rows == src->rows)      &&             
             (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(double));     
}